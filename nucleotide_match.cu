#include "hip/hip_runtime.h"
#include <stdio.h>

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <fstream>
#include <unistd.h>
#include <limits.h>
#include <string>

#include <vector>
#include <ctype.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/inner_product.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#define THREADS_PER_BLOCK	32
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

using namespace std;

static void HandleError( hipError_t err,const char *file, int line);

__global__ void create_number(char* sequence,
	 		uint32_t* number_values,
			uint32_t max_gene_length,
			uint32_t search_sequence_length,
			uint32_t total_numbers);

uint32_t* get_threadsPerBlock_numberConversion (uint32_t max_gene_length,
						uint32_t search_sequence_length,
						uint32_t total_numbers,
						uint32_t total_gene_sequences);

__global__ void get_count (uint32_t* gene_sequence_numbers,
	 		uint32_t* search_number_values,
			uint16_t* count,
			uint32_t total_gene_sequences,
			uint32_t total_search_sequences,
			uint32_t numbers_per_gene);

//************************************************************************************************

__global__ void get_count (uint32_t* gene_sequence_numbers,
	 		uint32_t* search_number_values,
			uint16_t* count,
			uint32_t total_gene_sequences,
			uint32_t total_search_sequences,
			uint32_t numbers_per_gene)
{

	uint32_t blockId = blockIdx.y * gridDim.x + blockIdx.x;	
	uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;
	uint32_t id = blockId * blockDim.x + threadIdx.x;

	uint32_t i=0;
	uint16_t count_value;

	if(idx < total_search_sequences) {

		count_value = 0;

		for(i=0; i<numbers_per_gene; i++) {
			//printf("Search Number is: %d\n", search_number_values[idx]);
			if(search_number_values[idx] == gene_sequence_numbers[idy * numbers_per_gene + i]) {
				count_value++;
			}
		}

		count[idx * total_gene_sequences + idy] = count_value;
/*
		if((idy == 0) || (idy == 1)) {
			printf("Total Search Sequences: %d Idx: %d Idy: %d Sequence: %d Count: %d\n", total_search_sequences,idx, idy, search_number_values[idx], count[idx*total_gene_sequences + idy]);
		}*/
	}

}


//----------------------------------------------------------------------------------------------//
__global__ void create_number(char* gene_sequence,
	 		uint32_t* number_values,
			uint32_t max_gene_length,
			uint32_t search_sequence_length,
			uint32_t total_numbers)
{

	uint32_t blockId = blockIdx.y * gridDim.x + blockIdx.x;
	uint32_t id = blockId * blockDim.x + threadIdx.x;
	uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;


	//if(idx < (max_gene_length - search_sequence_length + 1)) {
	if(idx < total_numbers) {

		//char* string;
		uint32_t number = 0;


		uint16_t i,j;
		uint8_t k;
		j=id;
		k=0;


		uint32_t multiplier = 1;


		//pow() function does not give exact result
		//This is basically getting pow(10, search_sequence_length)
		for(i=1; i< search_sequence_length; i++) {
			multiplier *= 10;
		}


		for(i=0; i < search_sequence_length; i++, j++) {

			//printf("Number at start is: %d\n",number);

		//for(i=idx; i<(idx+search_sequence_length); i++,j++,k++)

			switch(gene_sequence[j]) {
				case ('A'):
					//string[i] = '1';
					number+= 1 * multiplier;
					break;
				case ('T'):
					//string[i] = '2';
					number+= 2 * multiplier;
					break;
				case ('G'):
					//string[i] = '3';
					number+= 3 * multiplier;
					break;
				case ('C'):
					//string[i] = '4';
					number+= 4 * multiplier;
					break;
				case ('a'):
					//string[i] = '1';
					number+= 1 * multiplier;
					break;
				case ('t'):
					//string[i] = '1';
					number+= 2 * multiplier;
					break;
				case ('g'):
					//string[i] = '1';
					number+= 3 * multiplier;
					break;
				case ('c'):
					//string[i] = '1';
					number+= 4 * multiplier;
					break;
				case ('N'):
					//string[i] = '7';
					number+= 7 * multiplier;
					break;
				case ('n'):
					//string[i] = '7';
					number+= 7 * multiplier;
					break;
				case ('\0'):
					//string[i] = '9';
					number+= 9 * multiplier;
					break;
				default:
					number+= 9 * multiplier;
			}

			multiplier = multiplier / 10;

			//if(idx == 0) {
				//printf("Number is: %d \t Multiplier value is: %d\n", number, multiplier );
			//}
		}
		//scanf(string,"%d",&number);
		//number = atoi(string);


		number_values[id] = number;
		//printf("Total Numbers is: %d BlockDim.x: %d Thread Y ID: %d Thread X Id: %d Number is: %d\n",total_numbers, blockDim.x, idy,idx,number);
	}
}


template <typename Vector1, typename Vector2>
void dense_histogram(const Vector1& input, Vector2& histogram)
{

	typedef typename Vector1::value_type ValueType; // input value type
	typedef typename Vector2::value_type IndexType; // histogram index type

	// copy input data (could be skipped if input is allowed to be modified)
	thrust::device_vector<ValueType> data(input);

	  // print the initial data
  	print_vector("initial data", data);


	// sort data to bring equal elements together
	thrust::sort(data.begin(), data.end());

	// number of histogram bins is equal to the maximum value plus one
	IndexType num_bins = data.back() + 1;

	// resize histogram storage
	histogram.resize(num_bins);

	// find the end of each bin of values
	thrust::counting_iterator<IndexType> search_begin(0);
	thrust::upper_bound(data.begin(), data.end(),
		      search_begin, search_begin + num_bins,
		      histogram.begin());

	// print the cumulative histogram
	print_vector("cumulative histogram", histogram);

	// compute the histogram by taking differences of the cumulative histogram
	thrust::adjacent_difference(histogram.begin(), histogram.end(),
		              histogram.begin());

	// print the histogram
	print_vector("histogram", histogram);
}



template <typename Vector>
void print_vector(const std::string& name, const Vector& v)
{
	typedef typename Vector::value_type T;

	std::cout << "  " << std::setw(20) << name << "  ";

	thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(std::cout, " "));

	std::cout << std::endl;
}


static void HandleError( hipError_t err,const char *file, int line)
{
	if (err != hipSuccess)
	{
	        fprintf( stderr,"%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit(err);
    	}
}

//Function that gets the total threads per block required for the number calculation kernel
uint32_t* get_threadsPerBlock_numberConversion (uint32_t max_gene_length,
						uint32_t search_sequence_length,
						uint32_t total_numbers,
						uint32_t total_gene_sequences)
{
	uint32_t threadsPBlock;
	uint32_t blocks_x;
	static uint32_t returnArr[3];

	if(max_gene_length > 1024) {
		threadsPBlock = 1024;
		blocks_x = (max_gene_length + 1023) / 1024;
	}
	else if ((max_gene_length < 1024) && (max_gene_length > 512)) {
		threadsPBlock = 512;
		blocks_x = (max_gene_length + 511) / 512;
	}
	else {
		threadsPBlock = 32;
		blocks_x = (max_gene_length + 31) / 32;
	}

	returnArr[0] = threadsPBlock;
	returnArr[1] = blocks_x;
	returnArr[2] = total_gene_sequences;

	return returnArr;

}

int main(int argc, char* argv[])
{

	/**** User supplied variables *****/

	//Total number of gene sequences in the file
	uint32_t total_gene_sequences;

	//Maximum length of each gene sequence
	uint32_t max_gene_length;

	uint32_t total_search_sequences;

	uint32_t search_sequence_length;





	/**** Host TypeData Arrays  *********/

	//vector<vector<string> > gene_sequences;
	//vector<string> search_sequences;

	//Search sequences in 4 byte integers
	uint32_t * search_sequence_numbers;
	char* gene_sequences;
	uint16_t* count;


	/**** Device TypeData Arrays  *********/

	char* d_gene_sequences;
	uint32_t* d_search_sequence_numbers;
	uint32_t* d_gene_number_sequences;
	uint16_t* d_count;

	/***************  Temporary Variables **********************/

	//Temporary variables needed for loops
	int i,j;

	//Variable needed for getopt; Holds the the option; In "-n 10" holds 'n' and 'optarg' holds 10
	int option;


	/***************  Reading the user arguments ***************/


	//Checking if correct number of arguments were provided
	if(argc != 7){
		printf("Usage: Executable -n Number_of_Genes -m Max_Gene_Length -l Search_Sequence_Length \n");
	 	exit(EXIT_FAILURE);
	}

	//Parsing the options provided
	while((option = getopt(argc,argv,"n:m:l:")) != -1){
		switch (option){

			case 'n':
				total_gene_sequences = atoi(optarg);
				break;
			case 'm':
				max_gene_length = atoi(optarg);
				break;
			case 'l':
				search_sequence_length = atoi(optarg);
				break;
			default:
				printf("Usage: Executable -n Number_of_Genes -m Max_Gene_Length -l Search_Sequence_Length \n");
			 	exit(EXIT_FAILURE);
		}
	}



	//Total number of sequences of size search_sequence_length in each gene
	uint32_t numbers_per_gene = max_gene_length - search_sequence_length + 1;


	/***************  Reading input data from file **********/

/*
	//----------------Reading all genes------------//
	ifstream gene_file("fasta_1.txt");

	string line;

	i=0;

	//Temporary vector to push onto the gene_sequences vector
	vector<string> temp_vector;

	while(getline(gene_file, line)) {

		if(line.at(0) == '>') {
			continue;
		}
		else {
			temp_vector.clear();
			temp_vector.push_back(line);
			gene_sequences.push_back(temp_vector);

		}
		i++;
	}

	cout << "Vector size is: " << gene_sequences.size() << "\n";

*/
	printf("Max Gene Length: %d\n",max_gene_length);

	//Getting the total number of permutations; 4 ^ length of the search sequence
	total_search_sequences = 2 << (search_sequence_length* 2 - 1);



	/***************  Allocating memoery for host arrays *******/

	search_sequence_numbers = (uint32_t*)malloc(total_search_sequences * sizeof(uint32_t));
	gene_sequences = (char*)malloc(total_gene_sequences * max_gene_length * sizeof(char));
	count = (uint16_t*)malloc(total_gene_sequences * total_search_sequences * sizeof(uint32_t)); 

	/***************  Allocating memoery for device arrays *****/

	HANDLE_ERROR(hipMalloc((void**)&d_search_sequence_numbers, total_search_sequences * sizeof(uint32_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_gene_sequences, total_gene_sequences * max_gene_length * sizeof(char)));
	HANDLE_ERROR(hipMalloc((void**)&d_gene_number_sequences, numbers_per_gene * total_gene_sequences * sizeof(uint32_t)));

	/***************  Reading the files for input data *********/

	//Open file where A, T, G, C is replaced by 1, 2, 3 and 4 respectively
	FILE * search_sequences_file;
	FILE * gene_sequences_file;

	search_sequences_file = fopen("all_size_eight_combinations_numbers.txt","r");
	gene_sequences_file = fopen("fasta_test_10_seqs.txt","r");

	char *line;
	char *line2;
	uint32_t integer_value;


	//-----------Reading all combinations----------//
	i=0;

	line = (char*)malloc((search_sequence_length + 10) * sizeof(char));

	while(fgets(line, search_sequence_length + 10, search_sequences_file)) {

		integer_value = atof(line);
		search_sequence_numbers[i] = integer_value;

		i++;
	}

	fclose(search_sequences_file);
	free(line);

	//-----------Reading all genes ----------------//

	//memset(gene_sequences,'0',sizeof(gene_sequences));

	i=0;
	j=0;

	line2 = (char*)malloc((max_gene_length + 2) * sizeof(char));

	memset(line,'\0',max_gene_length + 2);

	while(fgets(line2, max_gene_length + 2, gene_sequences_file)) {


		if(line2[0] == '>') {
			continue;
		}
		//printf("%s\n",line2);
		strncpy(gene_sequences + i,line2,max_gene_length);
		//printf("%s\n",(gene_sequences + i));
/*		for(j=0; j < max_gene_length; j++, i++) {
			gene_sequences[i] = line2[j];
			printf("%c",gene_sequences[j]);
		}
*/
		i+=max_gene_length;
		memset(line,'\0',max_gene_length + 2);
	}


	fclose(gene_sequences_file);


	free(line2);


	/***************  Copying data from Host to Device *****/

	HANDLE_ERROR(hipMemcpy(d_search_sequence_numbers, search_sequence_numbers, total_search_sequences * sizeof(uint32_t) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_gene_sequences, gene_sequences, total_gene_sequences * max_gene_length * sizeof(char) , hipMemcpyHostToDevice));



	uint32_t* blockGridDims = get_threadsPerBlock_numberConversion (max_gene_length, search_sequence_length, numbers_per_gene, total_gene_sequences);
	uint32_t threads_per_block = blockGridDims[0];
	uint32_t gridSize_x = blockGridDims[1];
	uint32_t gridSize_y = blockGridDims[2];

	printf("Threads/Block, X GridSize, Y GridSize: %d %d %d\n",threads_per_block, gridSize_x, gridSize_y);


	dim3 blockSize(threads_per_block,1,1);
	dim3 gridSize(gridSize_x, gridSize_y,1);


//	dim3 blockSize(threads_per_block,1,1);
//	dim3 gridSize(3,1,1);


	create_number<<<gridSize, blockSize>>>(d_gene_sequences, d_gene_number_sequences, max_gene_length, search_sequence_length, numbers_per_gene);
	HANDLE_ERROR(hipDeviceSynchronize());

	//Freeing memory no longer required for calculations
	HANDLE_ERROR(hipFree(d_gene_sequences));
	free(gene_sequences);

	//---------------------------------------------//
	
	//Allocating memory for the device array that will host the count values
	HANDLE_ERROR(hipMalloc((void**)&d_count, total_search_sequences * total_gene_sequences * sizeof(uint16_t)));
	HANDLE_ERROR(hipMemset((void *)d_count, 0, total_search_sequences * total_gene_sequences * sizeof(uint16_t)));
	




	dim3 blockSize2(512,1,1);
	dim3 gridSize2(128,total_gene_sequences,1);

	get_count <<<gridSize2, blockSize2>>>(d_gene_number_sequences, d_search_sequence_numbers, d_count, total_gene_sequences, total_search_sequences,numbers_per_gene);
	HANDLE_ERROR(hipDeviceSynchronize());

	HANDLE_ERROR(hipMemcpy(count, d_count, total_gene_sequences * total_search_sequences * sizeof(uint16_t), hipMemcpyDeviceToHost));



	//----------------------------------------------//

	for(i=0;i<total_search_sequences; i++) {

		printf("%d ",search_sequence_numbers[i]);

		for(j=0; j<total_gene_sequences; j++) {
			//count[idx*total_gene_sequences + idy]
			printf("%d ",count[i * total_gene_sequences + j]);
			//printf("%d ",(uint32_t)count[i * total_gene_sequences + j]);
		}
		printf("\n");

	}


	//-----------Reading all combinations----------//
/*
	ifstream combination_file("all_size_eight_combinations.txt");

	string line_search;

	while(getline(combination_file, line_search)) {
		search_sequences.push_back(line_search);
	}

	cout << "Search Vector size is: " << search_sequences.size() << "\n";
	cout<< search_sequences.at(0) << "\n";


	//Creating numbers for each 8 sequence
	uint32_t* number_values;
*/
	//1D Grid with blocksize containing 32 threads
/*	dim3 gridSize(((totalObservations*totalTypes) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1,1);
	dim3 blockSize(32,1,1);

	//calculate_number<<<>>>(gene_string,number_values);
*/
	/***************  Generating Histogram ***************/
/*
	thrust::device_vector<int> histogram;
	thrust:: device_vector <string> vec1;

	vec1 = gene_sequences.at(0);

	print_vector("Test", vec1);
*/
	//dense_histogram(vec1, histogram);


	HANDLE_ERROR(hipFree(d_search_sequence_numbers));

	HANDLE_ERROR(hipFree(d_gene_number_sequences));
	HANDLE_ERROR(hipFree(d_count));

	free(count);
	free(search_sequence_numbers);

}
